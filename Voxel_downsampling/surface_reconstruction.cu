#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  

__global__
void replace_idx(int* d_surface, int idx_change, int idx_replace)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (d_surface[tid * 3 + 0] == idx_change) d_surface[tid * 3 + 0] = idx_replace;
	if (d_surface[tid * 3 + 1] == idx_change) d_surface[tid * 3 + 1] = idx_replace;
	if (d_surface[tid * 3 + 1] == idx_change) d_surface[tid * 3 + 2] = idx_replace;
}

//----FIRST PASS----
//Group connections from all points in a voxel
__global__
void first_pass(float* input_cloud, int* surface, int* idx_points, int* pos_out, int* repeat, int num_points_out, int num_points)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int block_size = 1024, grid_size = num_points / block_size + 1;//for replacement
	int idx, idx_in_voxel;
	float centroid[3] = {};

	//take all valid points and with repeatitions greater than 1
	if (tid < num_points_out && repeat[tid] > 1)
	{
		//get the first index of the voxel
		idx = idx_points[pos_out[tid]];
		centroid[0] = 0; centroid[1] = 0; centroid[2] = 0;
		for (int i = 1; i < repeat[tid]; i++)
		{
			idx_in_voxel = idx_points[pos_out[tid] + i];
			replace_idx <<< grid_size, block_size >>> (surface, idx_in_voxel, idx);
			centroid[0] += input_cloud[idx_in_voxel * 3 + 0];
			centroid[1] += input_cloud[idx_in_voxel * 3 + 1];
			centroid[2] += input_cloud[idx_in_voxel * 3 + 2];
		}
		input_cloud[idx * 3 + 0] = centroid[0] / repeat[tid];
		input_cloud[idx * 3 + 1] = centroid[1] / repeat[tid];
		input_cloud[idx * 3 + 2] = centroid[2] / repeat[tid];
	}
	//__syncthreads();
}

__global__
void second_pass()
{

}

int generate_surface_reconstruction(int* d_surface, float* d_input_cloud, int* h_idx_points, int* h_pos_out, int* h_repeat, int num_points_out)
{
	// allocate memory on GPU
	int* d_pos_out, * d_repeat;
	size_t bytes_out_int = (size_t)num_points_out * sizeof(int);
	checkCudaErrors(hipMalloc(&d_pos_out, bytes_out_int));
	checkCudaErrors(hipMalloc(&d_repeat, bytes_out_int));

	// transfer data from CPU to GPU
	checkCudaErrors(hipMemcpy(d_pos_out, h_pos_out, bytes_out_int, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_repeat, h_repeat, bytes_out_int, hipMemcpyHostToDevice));

	// number of threads
	int block_size = 1024, grid_size = num_points_out / block_size + 1;
	printf("\nBlock size: %d\n", block_size);
	printf("Grid size: %d\n", grid_size);
	printf("Total number of threads: %d\n", block_size * grid_size);
	hipError_t err;

	first_pass <<< grid_size, block_size >>> ();
}