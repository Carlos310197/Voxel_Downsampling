#include "hip/hip_runtime.h"
/*
Surface Reconstruction for any OS1 LiDAR based on voxel downsampling of a point cloud
Author: Carlos Huapaya
*/

//Input: Non-overlapped point cloud and its resulting surface reconstruction arrays
//Output: Surface reconstruction of the given point cloud based on the voxel downsampling algorithm (DXF format)

///NOTES:
//The size of the voxels has to be a user input (for now).
//There is a correspondence between the size of the voxels and the form of the scanned scene.
//The greater the voxels, the more information loss of the scene and so the less number of triangles (lighter DXF file)

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <time.h>

#define LEAF_SIZE 800.0f

float* read_point_cloud(const char* name, int* num_points);
int* read_surface(const char* name, int* num_points);
float* voxel_downsampling(float* h_input_cloud, float* d_input_cloud, int num_points, float* h_leaf_size, int** h_idx_points, int** h_idx_voxels, int** h_pos_out, int** h_repeat, int* num_points_out);
int* generate_surface_reconstruction(int* h_surface, int* h_idx_points, int* h_pos_out, int* h_repeat, int num_points_out, int num_points, int num_triangles, int* num_triangles_out);

int main()
{
	int n_donuts = 6;//number of donuts to process
	const char sphere_name[] = "point_cloud_mine.csv";//name of the input cloud
	const char surface_name[] = "surface.csv";//name of the input cloud
	int num_points = 0, num_triangles = 0;//initialize the number of points and triangles
	int num_points_out = 0, num_triangles_out = 0;//reduced number of points and triangles
	//hipError_t err, cudaStatus;
	clock_t start, end;
	long double time;

	//------------------------------------------
	//-----------Read the point cloud-----------
	//------------------------------------------
	float* h_sphere_pc = read_point_cloud(sphere_name, &num_points);
	if (h_sphere_pc == NULL) return -1;//check if there was any errors
	printf("Number of points read: %d\n", num_points);

	//allocate memory for the point cloud in the GPU
	float* d_sphere_pc;
	size_t bytes_sphere = (size_t)3 * (size_t)num_points * sizeof(float);
	checkCudaErrors(hipMalloc(&d_sphere_pc, bytes_sphere));

	//transfer the point cloud from the CPU to GPU
	checkCudaErrors(hipMemcpy(d_sphere_pc, h_sphere_pc, bytes_sphere, hipMemcpyHostToDevice));

	//------------------------------------------
	//-------------Read the surface-------------
	//------------------------------------------
	int* h_surface = read_surface(surface_name, &num_triangles);
	if (h_surface == NULL) return -1;//check if there was any errors
	printf("Number of triangles read: %d\n", num_triangles);

	int* h_surface_new = (int*)malloc((size_t)num_triangles * (size_t)3 * sizeof(int));
	printf("Size of h_surface: %d\n", sizeof(h_surface) / sizeof(int));

	//------------------------------------------
	//-----Create the voxel grid structure------
	//------------------------------------------
	printf("\n-----------Voxel Structure-----------\n");
	float h_leaf_size[3] = { LEAF_SIZE, LEAF_SIZE, LEAF_SIZE };// size of voxel
	int* h_idx_points = nullptr, * h_idx_voxels = nullptr, * h_pos_out = nullptr, * h_repeat = nullptr;

	// generate the voxel grid structure
	start = clock();
	float* downsampled_cloud = voxel_downsampling(h_sphere_pc, d_sphere_pc, num_points, h_leaf_size, &h_idx_points, &h_idx_voxels, &h_pos_out, &h_repeat, &num_points_out);
	end = clock();
	time = (double)(end - start) / (double)(CLOCKS_PER_SEC) * 1e3;
	printf("Elapsed time voxel structure: %.4lf ms\n", time);

	//------------------------------------------
	//----Compute the surface reconstruction----
	//------------------------------------------
	printf("\n---------Surface Reconstruction---------\n");
	start = clock();
	int* downsampled_surface = generate_surface_reconstruction(h_surface, h_idx_points, h_pos_out, h_repeat, num_points_out, num_points, num_triangles, &num_triangles_out);
	end = clock();
	time = (double)(end - start) / (double)(CLOCKS_PER_SEC) * 1e3;
	printf("Elapsed time surface reconstruction: %.4lf ms\n", time);

	//------------------------------------------
	//----------Generate the DXF file-----------
	//------------------------------------------
	//copy this from "Reconstruccion_tuberia"


	free(h_idx_points), free(h_idx_voxels), free(h_pos_out), free(h_repeat);
	free(h_sphere_pc), free(h_surface), free(downsampled_surface);

	return 0;
}

float* read_point_cloud(const char* name, int* num_points)
{
	//initialize memory for the point cloud with 3 points
	size_t pc_bytes = (size_t)(3) * 3 * sizeof(float);
	size_t new_pc_bytes;
	float* point_cloud = (float*)malloc(pc_bytes);
	if (!point_cloud) { printf("Error allocating memory for point cloud\n"); return NULL; }

	//read from the file
	const int N_LINE = 2048;
	FILE* document;
	fopen_s(&document, name, "r");//open the CSV document
	if (!document) { printf("File opening failed\n"); return NULL; }
	char line[N_LINE]; //pointer to the string in each line
	char* token = NULL;
	char sep[] = ",\n"; //space separation
	char* next_token = NULL;
	char* next_ptr = NULL;

	fgets(line, N_LINE, document);//read header

	//the cloud is stored using column-major format
	int i = 0;
	*num_points = 0;
	while (fgets(line, N_LINE, document) != NULL)
	{
		new_pc_bytes = (size_t)(3) * ((size_t)i + 1) * sizeof(float);
		if (i > 0) point_cloud = (float*)realloc(point_cloud, new_pc_bytes);//reallocate memory
		if (!point_cloud) { printf("Error allocating memory for point cloud\n"); return NULL; }
		token = strtok_s(line, sep, &next_token);
		while (token != NULL)//on the line
		{
			point_cloud[i] = strtof(token, &next_ptr);//convert from string to float
			token = strtok_s(NULL, sep, &next_token);//read next string
			i++;
		}
		(*num_points)++;
	}

	fclose(document);//close the document
	return point_cloud;
}

int* read_surface(const char* name, int* num_points)
{
	//initialize memory for the point cloud with 3 points
	size_t pc_bytes = (size_t)(3) * 3 * sizeof(int);
	size_t new_pc_bytes;
	int* surface = (int*)malloc(pc_bytes);
	if (!surface) { printf("Error allocating memory for point cloud\n"); return NULL; }

	//read from the file
	const int N_LINE = 2048;
	FILE* document;
	fopen_s(&document, name, "r");//open the CSV document
	if (!document) { printf("File opening failed\n"); return NULL; }
	char line[N_LINE]; //pointer to the string in each line
	char* token = NULL;
	char sep[] = ",\n"; //space separation
	char* next_token = NULL;
	char* next_ptr = NULL;

	fgets(line, N_LINE, document);//read header

	//the cloud is stored using column-major format
	int i = 0;
	*num_points = 0;
	while (fgets(line, N_LINE, document) != NULL)
	{
		new_pc_bytes = (size_t)(3) * ((size_t)i + 1) * sizeof(int);
		if (i > 0) surface = (int*)realloc(surface, new_pc_bytes);//reallocate memory
		if (!surface) { printf("Error allocating memory for surface\n"); return NULL; }
		token = strtok_s(line, sep, &next_token);
		while (token != NULL)//on the line
		{
			surface[i] = (int)strtof(token, &next_ptr);//convert from string to float
			token = strtok_s(NULL, sep, &next_token);//read next string
			i++;
		}
		(*num_points)++;
	}

	fclose(document);//close the document
	return surface;
}